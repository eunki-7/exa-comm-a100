/*
 * CUDA-aware MPI AllReduce example
 * - Allocates a GPU buffer, initializes with rank value,
 *   performs MPI_Allreduce (SUM) in-place on device memory,
 *   prints a few sample values from rank 0.
 * - Demonstrates GPUDirect RDMA path when supported (no CPU staging).
 */
#include <mpi.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <vector>

#define CHECK_MPI(x) do { int e=(x); if(e!=MPI_SUCCESS){ fprintf(stderr,"MPI error %d\n",e); MPI_Abort(MPI_COMM_WORLD,e);} } while(0)
#define CHECK_CUDA(x) do { hipError_t e=(x); if(e!=hipSuccess){ fprintf(stderr,"CUDA error %s\n",hipGetErrorString(e)); MPI_Abort(MPI_COMM_WORLD,1);} } while(0)

int main(int argc, char** argv){
    CHECK_MPI(MPI_Init(&argc, &argv));
    int rank,size; CHECK_MPI(MPI_Comm_rank(MPI_COMM_WORLD,&rank)); CHECK_MPI(MPI_Comm_size(MPI_COMM_WORLD,&size));

    int devCount=0; CHECK_CUDA(hipGetDeviceCount(&devCount));
    int dev = rank % (devCount>0?devCount:1);
    CHECK_CUDA(hipSetDevice(dev));

    const int N = 1<<20; // 1M floats (~4MB)
    float *d_buf=nullptr;
    CHECK_CUDA(hipMalloc(&d_buf, N*sizeof(float)));

    // initialize: fill with 'rank'
    std::vector<float> h_tmp(N, (float)rank);
    CHECK_CUDA(hipMemcpy(d_buf, h_tmp.data(), N*sizeof(float), hipMemcpyHostToDevice));

    // Allreduce directly on GPU memory (CUDA-aware MPI)
    CHECK_MPI(MPI_Allreduce(MPI_IN_PLACE, d_buf, N, MPI_FLOAT, MPI_SUM, MPI_COMM_WORLD));

    if(rank==0){
        std::vector<float> out(4);
        CHECK_CUDA(hipMemcpy(out.data(), d_buf, 4*sizeof(float), hipMemcpyDeviceToHost));
        printf("[rank0] sample: %f %f %f %f | expected sum 0..%d = %d\n",
               out[0], out[1], out[2], out[3], size-1, (size*(size-1))/2);
    }

    CHECK_CUDA(hipFree(d_buf));
    MPI_Finalize();
    return 0;
}
